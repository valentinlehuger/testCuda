#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>



__global__
void incrementOne(const int* tab,
                       int* newTab)
{
    int x = blockIdx.x;
    newTab[x] = tab[x] + 1;
}


void			incrementOne_cu(const int *h_tab,
								int ** d_tab,
								int ** d_newTab,
								int size)
{

	checkCudaErrors(hipMalloc(d_tab, sizeof(int) * size));
	checkCudaErrors(hipMalloc(d_newTab, sizeof(int) * size));
	checkCudaErrors(hipMemset(*d_newTab, 0, sizeof(int) * size));
	checkCudaErrors(hipMemcpy(*d_tab, h_tab, sizeof(int) * size, hipMemcpyHostToDevice));

	const dim3 blockSize(1, 1, 1);
	const dim3 gridSize(size, 1, 1);
	incrementOne<<<gridSize, blockSize>>>(*d_tab, *d_newTab);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

int main()
{
	return (0);
}