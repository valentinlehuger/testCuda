#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NB 10

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__global__ void max(float *d_in, float *d_out)
{
	int ft_id = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	int size = blockDim.x;

	// for (int i = 0; i < blockDim.x; i ++)
	// {
	// 	printf("%f\n", d_in[i]);
	// }

	for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			d_in[ft_id] = (d_in[ft_id] > d_in[ft_id + s]) ? d_in[ft_id] : d_in[ft_id + s];

			if (size % 2 == 1 && ft_id + s + s == size - 1)
				d_in[ft_id] = (d_in[ft_id] > d_in[ft_id + s + s]) ? d_in[ft_id] : d_in[ft_id + s + s];
		}
		__syncthreads();
		size /= 2;
	}
	if (tid == 0)
		d_out[blockIdx.x] = d_in[ft_id];
}

float			*init_values(size_t size)
{
	float		*values;

	if ((values = (float *)malloc(sizeof(float) * size)) != NULL)
	{
		for (size_t i = 0; i < size; i++)
			values[i] = float(i + 2);
		values[size / 2] = 100.f;
		return (values);
	}
	return (NULL);
}

int				main(void)
{
	float		*h_values;

	float		*d_values;
	float		**d_values_ = &d_values;

	float		*h_max = (float *)malloc(sizeof(float) * 1);
	float		*d_max;
	float		**d_max_ = &d_max;

	h_values = init_values(NB);

	printf ("Initial values :\n");
	for (int i = 0; i < NB; i++)
		printf("%f\n", h_values[i]);
	printf("\n");
	

	// malloc values and max
	checkCudaErrors(hipMalloc(d_values_, sizeof(float) * NB));
	checkCudaErrors(hipMalloc(d_max_, sizeof(float) * 1));

	// memcopy values
	checkCudaErrors(hipMemcpy(d_values, h_values, sizeof(float) * NB, hipMemcpyHostToDevice));

	// kernel
	max<<<1, NB>>>(d_values, d_max);

	// memcpy max result
	checkCudaErrors(hipMemcpy(h_max, d_max, sizeof(float) * 1, hipMemcpyDeviceToHost));

	// free the two
	hipFree(d_max_);
	hipFree(d_values_);

	printf("h_max = %f\n", *h_max);

	return(0);
}