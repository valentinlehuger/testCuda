#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define THREADS 4

#define ITEMS_PER_THREAD 30

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__global__ void	histogram_cuda(int *histogram, float *values, size_t nb, float bin_size, float min, int bins, int nb_thread)
{
  // nb = total size of elems


	int			id = (blockIdx.x * blockDim.x + threadIdx.x) * ITEMS_PER_THREAD;
	int			thread_id = threadIdx.x;
	int			*local_hist = (int *)malloc(sizeof(int) * bins);

	if (id == 0)
	  printf("Bin size : %f\n", bin_size);

	// Init local histogram
	for (int i = 0; i < bins; i++)
		local_hist[i] = 0;

	// One shared array per bin
	extern __shared__ int s_bins[];

	// Compute serially local bin
	for (int i = 0; i < ITEMS_PER_THREAD; i++)
	{
		for (int j = 0; j < bins; j += 1)
		{
			// if (id + i < NB)
			// 	printf("values[%d] = %f <= %f\n", id + i, values[id + i], (float)min + (float)(j + 1) * bin_size);

			if (id + i < nb && values[id + i] <= ((float)min + (float)(j + 1) * bin_size))
			{
				local_hist[j] += 1;
				//printf("BlockIdx : %d - Thread %d : values[%d] = %f -> local_hist[%d] = %d\n", blockIdx.x, thread_id, id + i, values[id + i], j, local_hist[j]);
				break ;
			}
		}
	}
	__syncthreads();
	// Store local bins into shared bins
	for (int i = 0; i < bins; i++)
	{
		s_bins[THREADS * i + thread_id] = local_hist[i];
		//		printf("Block %d - Thread %d : s_bins[%d] = local_hist[%d] = %d\n", blockIdx.x, thread_id, THREADS * i + thread_id, i, local_hist[i]);
	}

	__syncthreads();

	// if (thread_id == 0)
	// {
	// 	for (int i = 0; i < nb_thread * 3; i++)
	// 	{
	// 		printf("s_bins[%d] = %d\n", i, s_bins[i]);
	// 	}
	// }

	// Reduce each shared bin
	// int size = (blockIdx.x == gridDim.x - 1) ? (NB % blockDim.x) : blockDim.x;

	int size = THREADS;

	for (size_t s = THREADS / 2; s > 0; s >>= 1)
	{
		if (thread_id + s < THREADS && thread_id < s)
		{
			for (size_t j = 0; j < bins; j++)
			{
				s_bins[j * THREADS + thread_id] = s_bins[j * THREADS + thread_id] + s_bins[j * THREADS + thread_id + s];

				if (size % 2 == 1 && thread_id + s + s == size - 1)
					s_bins[j * THREADS + thread_id] = s_bins[j * THREADS + thread_id] + s_bins[j * THREADS + thread_id + s + s];
			}
		}
		__syncthreads();
		size = s;
	}

	// Store the result into histogram
	if (thread_id == 0)
	{
	  for (int i = 0; i < bins; i++) {
	    histogram[i + blockIdx.x * bins] = s_bins[THREADS * i];




	  //		histogram[0 + blockIdx.x * bins] = s_bins[0];
	  //		histogram[1 + blockIdx.x * bins] = s_bins[THREADS];
	  //		histogram[2 + blockIdx.x * bins] = s_bins[THREADS * 2];
	  //		printf("histogram[%d] = %d\n", 0 + blockIdx.x * bins, s_bins[0]);
	  //		printf("histogram[%d] = %d\n", 1 + blockIdx.x * bins, s_bins[THREADS]);
	  //  printf("histogram[%d] = %d\n", i + blockIdx.x * bins, s_bins[THREADS * i]);
	  }
	}
}


void			 histogram(float *h_values, size_t size, float min, float max, int *h_histogram, size_t bins)
{
	float		*d_values;
	float		**d_values_ = &d_values;

	int			*d_histogram;
	int			**d_histogram_ = &d_histogram;
	int			nb_thread = size / ITEMS_PER_THREAD + 1;


	int			grid_dim = nb_thread / THREADS + 1;

	// printf("size = %d\n", size);
	// printf("ITEMS_PER_THREAD = %d\n", ITEMS_PER_THREAD);
	// printf("nb_thread = %d\n", nb_thread);
	// printf("grid dim = %d\n", grid_dim);

	// cudaMalloc
	checkCudaErrors(hipMalloc(d_values_, sizeof(float) * size));
	checkCudaErrors(hipMalloc(d_histogram_, sizeof(int) * bins * grid_dim));

	// cudaMemcpy HostToDevice
	checkCudaErrors(hipMemcpy(d_values, h_values, sizeof(float) * size, hipMemcpyHostToDevice));

	// cudaMemset
	checkCudaErrors(hipMemset(d_histogram, 0, sizeof(int) * bins));

	

	// printf("size of shared mem = %d\n", THREADS * bins);
	//	printf("\n\n");
	// // kernel HISTOGRAM
	histogram_cuda<<<grid_dim, THREADS, THREADS * bins * sizeof(int) >>>(d_histogram, d_values, size, (float)(max - min) / (float)bins, min, bins, nb_thread);
	hipDeviceSynchronize();// checkCudaErrors(cudaGetLastError());


	// cudaMemcpy DeviceToHost
	checkCudaErrors(hipMemcpy(h_histogram, d_histogram, sizeof(int) * bins * grid_dim, hipMemcpyDeviceToHost));

	// cudaFree
	hipFree(d_values_);
	hipFree(d_histogram_);
}
