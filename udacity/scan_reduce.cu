#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}


#define NB 8


int			*init_values(size_t size)
{
	int		*values;

	if ((values = (int *)malloc(sizeof(int) * size)) != NULL)
	{
		for (size_t i = 0; i < size; i++)
			values[i] = i + 1;
		// values[size / 2] = 8.f;
		return (values);
	}
	return (NULL);
}

__global__ void	b_scan_reduce_cuda(int *values, int *cumulative)
{

	int		id = blockDim.x * blockIdx.x + threadIdx.x;
	int		tid = threadIdx.x;
////////// REDUCE
//
//
	int		size = blockDim.x;
	int		next_th = 2;
	printf("size = %u\n", size);
	for (int threshold = 1; threshold < blockDim.x / 2; threshold = threshold << 1)
	{
		// printf("Thread %d \t id = %d \t threshold = %d \t NB = 8 \t threadIdx.x = %d \t next_th = %d\n", tid, id, threshold, tid, next_th);
		printf("NB %d \t tid = %d \t next_th = %d\n", (NB - 1), tid, next_th);
		if (id < NB && id - threshold >= 0 && ((NB - 1) - tid) % next_th == 0)
		{
			printf("HERE : values[%d] = %d\n", id, values[id] + values[id - threshold]);
			values[id] = values[id] + values[id - threshold];
		}
		next_th = next_th << 1;
		__syncthreads();
	}

////////// DOWNSWEEP
//
//

// Store into cumulative

}

void		blelloch_scan_reduce(int *h_values, int *h_cumulative)
{
	int		*d_values;
	int		**d_values_ = &d_values;

	int		*d_cumulative;
	int		**d_cumulative_ = &d_cumulative;

	// mallocs
	checkCudaErrors(hipMalloc(d_values_, sizeof(int) * NB));
	checkCudaErrors(hipMalloc(d_cumulative_, sizeof(int) * NB));

	// memcpy & memset
	checkCudaErrors(hipMemcpy(d_values, h_values, sizeof(int) * NB, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_cumulative, h_values, sizeof(int) * NB, hipMemcpyHostToDevice));
	// checkCudaErrors(cudaMemset(d_cumulative, 0, sizeof(int) * NB));

	b_scan_reduce_cuda<<<1, 10>>>(d_values, d_cumulative);

	// memcpy
	checkCudaErrors(hipMemcpy(h_cumulative, d_values, sizeof(int) * NB, hipMemcpyDeviceToHost));
	// checkCudaErrors(cudaMemcpy(h_values, d_values, sizeof(int) * NB, cudaMemcpyDeviceToHost));

	// free
	hipFree(d_values_);
	hipFree(d_cumulative_);
}

int			main(void)
{
	int		*values;
	int		*cumulative;

	values = init_values(NB);
	cumulative = (int *)malloc(sizeof(int) * NB);
	blelloch_scan_reduce(values, cumulative);

	for (size_t i = 0; i < NB; i++)
	{
		printf("%d\t%d\n", values[i], cumulative[i]);
	}

	return (0);
}
